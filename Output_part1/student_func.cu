#include "hip/hip_runtime.h"
#include "reference_calc.cpp"
#include "utils.h"
#include <stdio.h>


__global__
void rgba_to_greyscale(const uchar4* const rgbaImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols)
{
        int i= (blockIdx.x*blockDim.x) + threadIdx.x; //x coordinate 
        int j= (blockIdx.y*blockDim.y) + threadIdx.y; //y cordinate
		int index=i+ j*numRows;
        uchar4 rgba = rgbaImage[index];//rgba values of the pixel to be processed 
        float value= (.299f*rgba.x) + (.587f*rgba.y) + (.114f*rgba.z); //formula
        greyImage[index] = (unsigned char)value; //storing the output value to the corresponding pixel of the grey image

}

void your_rgba_to_greyscale(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage,
                            unsigned char* const d_greyImage, size_t numRows, size_t numCols)
{
  int block_x= 32; // block size should always be a multiple of 32, because kernels issue instructions in warps (32 threads)
  int block_y= 1024/block_x;   // Assuming maximum active tread count 1024
  //block_y=((numCols/block_x)-1)<32?((numCols/block_x)-1):32;//**This was the best case which is runnng on 0.028736 but could not garantee for every case 
  const dim3 threadsPerBlock(block_x, block_y);  //assgin dimenctions for block
  const dim3 numBlocks( (numRows/block_x)+1, (numCols/block_y)+1); //assgin dimenctions for grid 
  rgba_to_greyscale<<<numBlocks, threadsPerBlock>>>(d_rgbaImage, d_greyImage, numRows, numCols);//kernel function call  
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}